#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int *X, int *Y, int a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  Y[i] += a * X[i];
}

int main() {
  int N = 10000000;
  int *X = (int*) malloc(sizeof(int) * N), 
      *Y = (int*) malloc(sizeof(int) * N),
       a = 2,
      *dX, *dY;
  
  hipMalloc(&dX, sizeof(int) * N);
  hipMalloc(&dY, sizeof(int) * N);

  for (int i = 0; i < N; i++) {
    X[i] = i;
    Y[i] = N - i;
  }

  hipMemcpy(dX, X, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dY, Y, sizeof(int) * N, hipMemcpyHostToDevice);

  dim3 blockDim(64, 1, 1);
  dim3 gridDim(N / blockDim.x, 1, 1);
  saxpy<<<gridDim, blockDim>>>(dX, dY, a);
  hipDeviceSynchronize();

  hipMemcpy(dY, Y, sizeof(int) * N, hipMemcpyDeviceToHost);

  free(X); free(Y);
  hipFree(dX); hipFree(dY);
}